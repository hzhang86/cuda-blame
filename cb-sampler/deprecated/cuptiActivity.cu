/*
 * Author: Hui Zhang
 * Modified based on Sangamesh Ragate's work
 * Date : 27th Sep 2017
 * UMD-CS
 * Description : This is the shared library that sets up the environent 
 * for the cuda application by creating the context and keeping it ready
 * to perform Exution Instrunction (sampling) of the cuda application as soon as it launces the kernel
 */



#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cupti.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

static CUpti_SubscriberHandle g_subscriber;
static FILE *samp_file;


#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define CUPTI_CALL(call)                                                      \
do {                                                                          \
    CUptiResult _status = call;                                               \
    if (_status != CUPTI_SUCCESS) {                                           \
        const char *errstr;                                                   \
        cuptiGetResultString(_status, &errstr);                               \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",  \
                __FILE__, __LINE__, #call, errstr);                           \
        exit(-1);                                                             \
    }                                                                         \
} while (0)

#define BUF_SIZE (32 * 16384)
#define ALIGN_SIZE (8)
#define ALIGN_BUFFER(buffer, align)                                            \
  (((uintptr_t) (buffer) & ((align)-1)) ? ((buffer) + (align) - ((uintptr_t) (buffer) & ((align)-1))) : (buffer)) 

/*
static char* stall_name[12];
static int val[12]={0};
    
static const char *
getStallReasonString(CUpti_ActivityPCSamplingStallReason reason,unsigned int samples)
{
    switch (reason) {
    case CUPTI_ACTIVITY_PC_SAMPLING_STALL_INVALID:
        stall_name[0]="Stall_invalid";
        val[0] += samples;
        return "Invalid";
    case CUPTI_ACTIVITY_PC_SAMPLING_STALL_NONE:
        stall_name[1]="Stall_none";
        val[1] += samples;
        return "Selected";
    case CUPTI_ACTIVITY_PC_SAMPLING_STALL_INST_FETCH:
        stall_name[2]="Stall_inst_fetch";
        val[2] += samples;
        return "Instruction fetch";
    case CUPTI_ACTIVITY_PC_SAMPLING_STALL_EXEC_DEPENDENCY:
        stall_name[3]="Stall_exec_dependency";
        val[3] += samples;
        return "Execution dependency";
    case CUPTI_ACTIVITY_PC_SAMPLING_STALL_MEMORY_DEPENDENCY:
        stall_name[4]="Stall_mem_dependency";
        val[4] += samples;
        return "Memory dependency";
    case CUPTI_ACTIVITY_PC_SAMPLING_STALL_TEXTURE:
        stall_name[5]="Stall_texture";
        val[5] += samples;
        return "Texture";
    case CUPTI_ACTIVITY_PC_SAMPLING_STALL_SYNC:
        stall_name[6]="Stall_sync";
        val[6] += samples;
        return "Sync";
    case CUPTI_ACTIVITY_PC_SAMPLING_STALL_CONSTANT_MEMORY_DEPENDENCY:
        stall_name[7]="Stall_const_mem_dependency";
        val[7] += samples;
        return "Constant memory dependency";
    case CUPTI_ACTIVITY_PC_SAMPLING_STALL_PIPE_BUSY:
        stall_name[8]="Stall_pipe_busy";
        val[8] += samples;
        return "Pipe busy";
    case CUPTI_ACTIVITY_PC_SAMPLING_STALL_MEMORY_THROTTLE:
        stall_name[9]="Stall_memory_throttle";
        val[9] += samples;
        return "Memory throttle";
    case CUPTI_ACTIVITY_PC_SAMPLING_STALL_NOT_SELECTED:
        stall_name[10]="Stall_warp_not_selected";
        val[10] += samples;
        return "Warp Not selected";
    case CUPTI_ACTIVITY_PC_SAMPLING_STALL_OTHER:
        stall_name[11]="Stall_other";
        val[11] += samples;
        return "Other";
    default:
        break;
    }

    return NULL;
}
*/


static void
printActivity(CUpti_Activity *record)
{
  switch (record->kind) {
    // The activity record for source locator contains the ID for the source path,
    // path for the file and the line number in the source.
    case CUPTI_ACTIVITY_KIND_SOURCE_LOCATOR:
    {
      CUpti_ActivitySourceLocator *sourceLocator = (CUpti_ActivitySourceLocator *)record;
      fprintf(samp_file, "SOURCE_LOCATOR SrcLctrId %d, File %s Line %d\n", sourceLocator->id, sourceLocator->fileName, sourceLocator->lineNumber);
      break;
    }
    // The activity record for instruction execution corresponds to a PC of the generated code, it contains the ID for source locator
    // the correlation ID of the kernel to which this record is associated, function ID and pc offset for the instruction.
    case CUPTI_ACTIVITY_KIND_INSTRUCTION_EXECUTION:
    {
      CUpti_ActivityInstructionExecution *sourceRecord = (CUpti_ActivityInstructionExecution *)record;
      fprintf(samp_file, "INSTRUCTION_EXECUTION srcLctr %u, corr %u, functionId %u, pc %x\n",
        sourceRecord->sourceLocatorId, sourceRecord->correlationId, sourceRecord->functionId, 
        sourceRecord->pcOffset);
      // number of threads that executed this instruction and number of times the instruction was executed
      fprintf(samp_file, "notPredOffthread_inst_executed %llu, thread_inst_executed %llu, inst_executed %u\n\n",
        (unsigned long long)sourceRecord->notPredOffThreadsExecuted, 
        (unsigned long long)sourceRecord->threadsExecuted, sourceRecord->executed);
      break;
    }
    // function name and corresponding module information
    case CUPTI_ACTIVITY_KIND_FUNCTION:
    {
      CUpti_ActivityFunction *fResult = (CUpti_ActivityFunction *)record;
      fprintf(samp_file, "FUCTION functionId %u, moduleId %u, name %s\n",
        fResult->id,
        fResult->moduleId,
        fResult->name);
      break;
    }
    // Kernel activity records kernel information
    case CUPTI_ACTIVITY_KIND_KERNEL:
    {
      CUpti_ActivityKernel3 *kernel = (CUpti_ActivityKernel3 *)record;
      fprintf(samp_file, "\n\n************************************** KERNEL_RECORD_SUMMARY **********************************\n");
      fprintf(samp_file, "Kernel %s , device %d, context %d, correlation %d, stream %d,[start-end][%ld-%ld]\n\n",kernel->name, 
             kernel->deviceId,kernel->contextId,kernel->correlationId,kernel->streamId,kernel->start,kernel->end);
      break;
    }

    default:
      fprintf(samp_file, "  <unknown>\n");
      break;
  }
}

static void CUPTIAPI
bufferRequested(uint8_t **buffer, size_t *size, size_t *maxNumRecords)
{
  uint8_t *b;

  *size = BUF_SIZE;
  b = (uint8_t *)malloc(*size + ALIGN_SIZE);
  if (*buffer == NULL) {
    printf("Error: out of memory\n");
    exit(-1);
  }

  *buffer = ALIGN_BUFFER(b, ALIGN_SIZE);
  *maxNumRecords = 0;
}

static void CUPTIAPI
bufferCompleted(hipCtx_t ctx, uint32_t streamId, uint8_t *buffer, size_t size, size_t validSize)
{
  CUptiResult status;
  CUpti_Activity *record = NULL;

  char host_name[128];
  gethostname(host_name, 127);
  samp_file = fopen(host_name, "a");

  do {
    status = cuptiActivityGetNextRecord(buffer, validSize, &record);
    if(status == CUPTI_SUCCESS) {
      printActivity(record);
    }
    else if (status == CUPTI_ERROR_MAX_LIMIT_REACHED) {
      break;
    }
    else {
      CUPTI_CALL(status);
    }
  } while (1);

  size_t dropped;
  CUPTI_CALL(cuptiActivityGetNumDroppedRecords(ctx, streamId, &dropped));
  if (dropped != 0) {
    printf("Dropped %u activity records\n", (unsigned int)dropped);
  }

  fclose(samp_file);
  free(buffer);
}

#define DUMP_CUBIN 1

void CUPTIAPI dumpCudaModule(CUpti_CallbackId cbid, void *resourceDescriptor)
{
#if DUMP_CUBIN
  const char *pCubin;
  size_t cubinSize;

  //dump the cubin at MODULE_LOADED_STARTING
  CUpti_ModuleResourceData *moduleResourceData = (CUpti_ModuleResourceData *)resourceDescriptor;
#endif
          
  if (cbid == CUPTI_CBID_RESOURCE_MODULE_LOADED) {
#if DUMP_CUBIN
    // You can use nvdisasm to dump the SASS from the cubin. 
    // Try nvdisasm -b -fun <function_id> sass_to_source.cubin
    pCubin = moduleResourceData->pCubin;
    cubinSize = moduleResourceData->cubinSize;
              
    FILE *cubin;
    cubin = fopen("sass_source_map.cubin", "wb");
    fwrite(pCubin, sizeof(uint8_t), cubinSize, cubin);
    fclose(cubin);
#endif
  }else if (cbid == CUPTI_CBID_RESOURCE_MODULE_UNLOAD_STARTING) {
    // You can dump the cubin either at MODULE_LOADED or MODULE_UNLOAD_STARTING
  }
}

static void
handleResource(CUpti_CallbackId cbid, const CUpti_ResourceData *resourceData)
{
  if (cbid == CUPTI_CBID_RESOURCE_MODULE_LOADED) {
    dumpCudaModule(cbid, resourceData->resourceDescriptor);
  }else if (cbid == CUPTI_CBID_RESOURCE_MODULE_UNLOAD_STARTING) {
    dumpCudaModule(cbid, resourceData->resourceDescriptor);
  }
}


static void CUPTIAPI
traceCallback(void *userdata, CUpti_CallbackDomain domain,
                      CUpti_CallbackId cbid, const void *cbdata)
{
  if (domain == CUPTI_CB_DOMAIN_RESOURCE) {
    handleResource(cbid, (CUpti_ResourceData *)cbdata);
  }
}


__attribute__((constructor)) void
initTrace()
{
  //get the arguments from the environment variables
  int deviceId;
  hipDeviceProp_t g_deviceProp;
  hipCtx_t cuCtx;

  printf("In initTrace \n");
  if (getenv("GPU_DEVICE_ID") != NULL) 
    deviceId = atoi(getenv("GPU_DEVICE_ID"));
  else deviceId = 0;

  RUNTIME_API_CALL(hipGetDeviceProperties(&g_deviceProp, deviceId));
  printf("Device Name: %s\n", g_deviceProp.name);
  if (g_deviceProp.major < 2) {
    printf("INSTRUCTION EXECUTION not supported on pre-Fermi devices\n");
    exit(-1);
  }
    
  hipInit(0);
  hipCtxCreate(&cuCtx,0,deviceId);
  CUPTI_CALL(cuptiActivityRegisterCallbacks(bufferRequested, bufferCompleted));
  CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_INSTRUCTION_EXECUTION));
  //CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_KERNEL));
  CUPTI_CALL(cuptiSubscribe(&g_subscriber, (CUpti_CallbackFunc)traceCallback, NULL));
  CUPTI_CALL(cuptiEnableDomain(1, g_subscriber, CUPTI_CB_DOMAIN_RESOURCE));
}

__attribute__((destructor)) void
finiTrace()
{
  CUPTI_CALL(cuptiActivityFlushAll(0));
  CUPTI_CALL(cuptiUnsubscribe(g_subscriber));
  CUPTI_CALL(cuptiActivityDisable(CUPTI_ACTIVITY_KIND_INSTRUCTION_EXECUTION));
  //CUPTI_CALL(cuptiActivityDisable(CUPTI_ACTIVITY_KIND_KERNEL));
  printf("In finiTrace \n"); //wait for all cupti api calls end
}

